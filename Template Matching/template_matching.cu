#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/imgcodecs/imgcodecs.hpp>

using namespace std;
using namespace cv;

__global__
void TMKernel(unsigned char * src, int width_src,
			  unsigned char * templ, int width_templ, int height_templ,
			  float * result, int width_result, int height_result)
{
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int Row = blockIdx.y * blockDim.y + threadIdx.y;

	if (Col >= width_result || Row >= height_result) return;

	float pixelVal = 0.0;
	float pixelTempSq = 0.0;
	float pixelSrcSq = 0.0;

	for (int templRow = 0; templRow < height_templ; templRow++)
	{
		for (int templCol = 0; templCol < width_templ; templCol++)
		{
			int curRow = Row + templRow;
			int curCol = Col + templCol;

			pixelVal += templ[templRow * width_templ + templCol] * 
						src[curRow * width_src + curCol];
			pixelTempSq += powf(templ[templRow * width_templ + templCol], 2);
			pixelSrcSq += powf(src[curRow * width_src + curCol], 2);
		}
	}

	result[Row * width_result + Col] = pixelVal / sqrtf(pixelTempSq * pixelSrcSq);
}

void TemplateMatchingWrapper(Mat &src, Mat &templ, Mat &result)
{
	unsigned char * dataImgSrc;
	unsigned char * dataImgTmpl;
	float * dataImgResult;

	int width_src = src.size().width;
	int height_src = src.size().height;

	int width_templ = templ.size().width;
	int height_templ = templ.size().height;

	int width_result = width_src - width_templ + 1;
	int height_result = height_src - height_templ + 1;

	int src_size = width_src * height_src * sizeof(unsigned char);
	int templ_size = width_templ * height_templ * sizeof(unsigned char);
	int result_size = width_result * height_result * sizeof(float);

	dataImgSrc = src.data;
	dataImgTmpl = templ.data;
	dataImgResult = (float *) malloc(result_size);

	unsigned char *d_Src, *d_Templ;
	float *d_Result;

	hipMalloc((void **) &d_Src, src_size);
	hipMalloc((void **) &d_Templ, templ_size);
	hipMalloc((void **) &d_Result, result_size);

	hipMemcpy(d_Src, dataImgSrc, src_size, hipMemcpyHostToDevice);
	hipMemcpy(d_Templ, dataImgTmpl, templ_size, hipMemcpyHostToDevice);

	dim3 dimGrid(ceil(width_result/32.0), ceil(height_result/32.0), 1);
	dim3 dimBlock(32, 32, 1);
	TMKernel<<<dimGrid, dimBlock>>>(d_Src, width_src, 
									d_Templ, width_templ, height_templ, 
									d_Result, width_result, height_result);

	hipMemcpy(dataImgResult, d_Result, result_size, hipMemcpyDeviceToHost);

	hipFree(d_Src); hipFree(d_Templ); hipFree(d_Result);

	result = Mat(height_result, width_result, CV_32FC1, dataImgResult);
}


int main()
{

	Mat img = imread("Technophilia.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	Mat templ = imread("template.jpg", CV_LOAD_IMAGE_GRAYSCALE);
	Mat result;

	TemplateMatchingWrapper(img, templ, result);

	double minVal; double maxVal; Point minLoc; Point maxLoc;

	minMaxLoc(result, &minVal, &maxVal, &minLoc, &maxLoc);

	rectangle(img, maxLoc, Point(maxLoc.x + templ.cols, maxLoc.y + templ.rows), Scalar::all(0), 1);
	imshow("Template Matching Result", result);
	imshow("Template Matching Found", img);
	waitKey(0);
	destroyAllWindows();

	return 0;
}